#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <cmath>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <stdio.h>

#define BLOCK_SIZE 32

__global__ void calc_dist(double* X, double* Y, double* Dist, int N) {
    int i = blockIdx.y*blockDim.y+threadIdx.y;
    int j = blockIdx.x*blockDim.x+threadIdx.x;

    // Essa matriz é simetrica, mas estamos calculando ela inteira. Ponto de otimizacao!
    Dist[i*N+j] = sqrt(pow((X[i] - X[j]), 2) + pow((Y[i] - Y[j]), 2));
}

__device__ void swap(int *a, int *b) {
    int temp = *a;
    *a = *b;
    *b = temp;
    return;
}

__global__ void random_sol(int *solutions, double *costs, double *distances, int N) {
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    double solution_cost = 0; // Custo total dessa solucao

    // Preenche a solucao em ordem para que possamos permutar depois
    for (int k = 0; k < N; k++) {
        solutions[i * N + k] = k;
    }
    
    // Inicializar o random
    hiprandState_t st;
    hiprand_init(0, i, 0, &st);
    int idx;

    // Realiza a permutacao e calcula o custo total da solucao
    for (int k = 1; k < N; k++){
        idx = (int) ((N-k) * hiprand_uniform(&st) + k); // Pegar um indice aleatorio entre 1 e N-1

        // Swap dos elementos do vetor e salva no vetor de solucoes
        swap(&solutions[i * N + k], &solutions[i * N + idx]);

        solution_cost += distances[solutions[i * N + k-1] * N + solutions[i * N + k]]; // Calculo das distancias
    }
    solution_cost += distances[solutions[i * N] * N + solutions[i * N + N-1]]; // Ultimo calculo: primeiro e ultimo

    costs[i] = solution_cost; // Salva no vetor de custos
}

int main() {
    // Preparacao para receber os dados do arquivo
    int N; std::cin >> N;
    thrust::host_vector<double> host_x(N);
    thrust::host_vector<double> host_y(N);

    double x, y;
    for (int i = 0; i < N; i++) {
        std::cin >> x; std::cin >> y;
        host_x[i] = x;
        host_y[i] = y;
    }
    // ---------------------------------------------------------------------
    // Preparacao para pre-calcular as distancias
    thrust::device_vector<double> dev_x(host_x);
    thrust::device_vector<double> dev_y(host_y);
    thrust::device_vector<double> dev_points_distance(N * N);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(ceil((double) N / threads.x), ceil((double) N / threads.y));

    calc_dist<<<grid,threads>>>(thrust::raw_pointer_cast(dev_x.data()), 
        thrust::raw_pointer_cast(dev_y.data()),
        thrust::raw_pointer_cast(dev_points_distance.data()), 
        N);

    // ---------------------------------------------------------------------
    // Preparacao sortear solucoes e calcular custos
    long nSols = 10000;
    int gpu_threads = 1024;
    
    thrust::device_vector<int> dev_solutions(nSols * N); // Vetor de solucoes
    thrust::device_vector<double> dev_costs(nSols); // Vetor de custos totais de cada solucao

    // Medicao de Tempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);
    
    random_sol<<<ceil((double) nSols/gpu_threads), gpu_threads>>>(thrust::raw_pointer_cast(dev_solutions.data()), 
        thrust::raw_pointer_cast(dev_costs.data()), 
        thrust::raw_pointer_cast(dev_points_distance.data()), 
        N);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    // ---------------------------------------------------------------------
    // Pegar o elemento minimo do vetor
    thrust::device_vector<double>::iterator iter = thrust::min_element(dev_costs.begin(), dev_costs.end());
    int position = iter - dev_costs.begin();
    double min_val = *iter;

    // ---------------------------------------------------------------------
    // Print do tempo e do melhor caminho
    #ifdef TIME
        std::cout << msecTotal << std::endl;
        std::cout << "milisegundo(s)." << std::endl;
    #endif

    std::cout << std::fixed << std::setprecision(5);
    std::cout << min_val;
    std::cout << " 0" << std::endl;

    for (int i = position * N; i < position * N + N; i++) {
        std::cout << dev_solutions[i] << ' ';
    }
    std::cout << std::endl;

    return 0;
}